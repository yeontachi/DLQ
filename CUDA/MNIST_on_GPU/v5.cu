#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// CORRECTED timing structure - separating actual operations
typedef struct {
    double memory_transfers;  // H2D + D2H only
    double gpu_compute;      // Forward + Backward + Update (GPU work)
    double host_computation; // Loss + grad computation only
    double total_time;
} TimingStats;

#define INPUT_SIZE 784
#define HIDDEN_SIZE 256
#define OUTPUT_SIZE 10
#define TRAIN_SIZE 10000
#define TEST_SIZE 10000
#define BATCH_SIZE 8
#define EPOCHS 10
#define LEARNING_RATE 0.01

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s (%d)\n", __FILE__, __LINE__, \
                    hipGetErrorString(error), error); \
            hipDeviceReset(); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

#define CUBLAS_CHECK(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            fprintf(stderr, "cuBLAS error at %s:%d: %d\n", __FILE__, __LINE__, status); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

double get_time_diff(struct timespec start, struct timespec end) {
    return (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;
}

typedef struct {
    float *d_weights1, *d_weights2, *d_bias1, *d_bias2;
    float *d_grad_weights1, *d_grad_weights2, *d_grad_bias1, *d_grad_bias2;
    float *d_fc1_output, *d_fc2_output, *d_grad_hidden, *d_grad_output;
    
    // PERSISTENT BUFFERS - NO MORE MALLOC/FREE PER BATCH
    float *d_input_batch;
    float *h_fc2_output;
    float *h_grad_output;
    
    hipblasHandle_t cublas_handle;
} NeuralNetworkCUDA;

void load_data(const char *filename, float *data, int size) {
    FILE *f = fopen(filename, "rb");
    if (!f) { perror("fopen data"); exit(EXIT_FAILURE); }
    fread(data, sizeof(float), size, f);
    fclose(f);
}

void load_labels(const char *filename, int *labels, int size) {
    FILE *f = fopen(filename, "rb");
    if (!f) { perror("fopen labels"); exit(EXIT_FAILURE); }
    fread(labels, sizeof(int), size, f);
    fclose(f);
}

void normalize_data(float *data, int size) {
    const float mean = 0.1307f;
    const float std = 0.3081f;
    for (int i = 0; i < size; i++) {
        data[i] = (data[i] - mean) / std;
    }
}

void initialize_weights_host(float *weights, int rows, int cols) {
    float scale = sqrtf(2.0f / rows);
    for (int i = 0; i < rows * cols; i++) {
        weights[i] = ((float)rand() / RAND_MAX) * 2.0f * scale - scale;
    }
}

void initialize_bias_host(float *bias, int size) {
    memset(bias, 0, size * sizeof(float));
}

__global__ void bias_add_kernel(float *x, float *bias, int batch, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch * size) {
        int bias_idx = idx % size;
        x[idx] += bias[bias_idx];
    }
}

__global__ void relu_kernel(float *x, int total) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total) {
        x[idx] = fmaxf(0.0f, x[idx]);
    }
}

__global__ void relu_backward_kernel(float *grad, float *x, int total) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total) {
        grad[idx] *= (x[idx] > 0.0f ? 1.0f : 0.0f);
    }
}

__global__ void bias_backward_kernel(float *grad_output, float *grad_bias, int batch, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch * size) {
        int bias_idx = idx % size;
        atomicAdd(&grad_bias[bias_idx], grad_output[idx]);
    }
}

// FORWARD PASS ONLY - separate function
void forward_pass_only(NeuralNetworkCUDA *nn, int batch_size) {
    const float alpha = 1.0f, beta = 0.0f;
    
    // Forward matmul 1: input * weights1
    CUBLAS_CHECK(hipblasSgemm(nn->cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                           HIDDEN_SIZE, batch_size, INPUT_SIZE,
                           &alpha, nn->d_weights1, HIDDEN_SIZE,
                           nn->d_input_batch, INPUT_SIZE, &beta,
                           nn->d_fc1_output, HIDDEN_SIZE));

    // Forward bias add 1
    int total_hidden = batch_size * HIDDEN_SIZE;
    int grid_hidden = (total_hidden + 255) / 256;
    bias_add_kernel<<<grid_hidden, 256>>>(nn->d_fc1_output, nn->d_bias1, batch_size, HIDDEN_SIZE);

    // Forward ReLU
    relu_kernel<<<grid_hidden, 256>>>(nn->d_fc1_output, total_hidden);

    // Forward matmul 2: hidden * weights2
    CUBLAS_CHECK(hipblasSgemm(nn->cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                           OUTPUT_SIZE, batch_size, HIDDEN_SIZE,
                           &alpha, nn->d_weights2, OUTPUT_SIZE,
                           nn->d_fc1_output, HIDDEN_SIZE, &beta,
                           nn->d_fc2_output, OUTPUT_SIZE));

    // Forward bias add 2 + SYNC (only because CPU needs this data)
    int total_out = batch_size * OUTPUT_SIZE;
    int grid_out = (total_out + 255) / 256;
    bias_add_kernel<<<grid_out, 256>>>(nn->d_fc2_output, nn->d_bias2, batch_size, OUTPUT_SIZE);
    CUDA_CHECK(hipDeviceSynchronize()); // Required for CPU copy
}

// BACKWARD PASS ONLY - separate function  
void backward_pass_only(NeuralNetworkCUDA *nn, int batch_size) {
    const float alpha = 1.0f, beta = 0.0f;
    
    // Zero gradients (async)
    CUDA_CHECK(hipMemset(nn->d_grad_weights1, 0, INPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMemset(nn->d_grad_weights2, 0, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMemset(nn->d_grad_bias1, 0, HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMemset(nn->d_grad_bias2, 0, OUTPUT_SIZE * sizeof(float)));

    // Backward matmul 2a: weights2 gradients
    CUBLAS_CHECK(hipblasSgemm(nn->cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                           OUTPUT_SIZE, HIDDEN_SIZE, batch_size,
                           &alpha, nn->d_grad_output, OUTPUT_SIZE,
                           nn->d_fc1_output, HIDDEN_SIZE, &beta,
                           nn->d_grad_weights2, OUTPUT_SIZE));

    // Backward bias2 gradients
    int total_out = batch_size * OUTPUT_SIZE;
    int grid_out = (total_out + 255) / 256;
    bias_backward_kernel<<<grid_out, 256>>>(nn->d_grad_output, nn->d_grad_bias2, batch_size, OUTPUT_SIZE);

    // Backward matmul 2b: hidden gradients
    CUBLAS_CHECK(hipblasSgemm(nn->cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                           HIDDEN_SIZE, batch_size, OUTPUT_SIZE,
                           &alpha, nn->d_weights2, OUTPUT_SIZE,
                           nn->d_grad_output, OUTPUT_SIZE, &beta,
                           nn->d_grad_hidden, HIDDEN_SIZE));

    // Backward ReLU
    int total_hidden = batch_size * HIDDEN_SIZE;
    int grid_hidden = (total_hidden + 255) / 256;
    relu_backward_kernel<<<grid_hidden, 256>>>(nn->d_grad_hidden, nn->d_fc1_output, total_hidden);

    // Backward matmul 1a: weights1 gradients
    CUBLAS_CHECK(hipblasSgemm(nn->cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                           HIDDEN_SIZE, INPUT_SIZE, batch_size,
                           &alpha, nn->d_grad_hidden, HIDDEN_SIZE,
                           nn->d_input_batch, INPUT_SIZE, &beta,
                           nn->d_grad_weights1, HIDDEN_SIZE));

    // Backward bias1 gradients
    bias_backward_kernel<<<grid_hidden, 256>>>(nn->d_grad_hidden, nn->d_grad_bias1, batch_size, HIDDEN_SIZE);
}

// WEIGHT UPDATES ONLY - separate function
void update_weights_only(NeuralNetworkCUDA *nn, float lr) {
    float neg_lr = -lr;
    
    CUBLAS_CHECK(hipblasSaxpy(nn->cublas_handle, INPUT_SIZE * HIDDEN_SIZE,
                           &neg_lr, nn->d_grad_weights1, 1, nn->d_weights1, 1));
    CUBLAS_CHECK(hipblasSaxpy(nn->cublas_handle, HIDDEN_SIZE * OUTPUT_SIZE,
                           &neg_lr, nn->d_grad_weights2, 1, nn->d_weights2, 1));
    CUBLAS_CHECK(hipblasSaxpy(nn->cublas_handle, HIDDEN_SIZE,
                           &neg_lr, nn->d_grad_bias1, 1, nn->d_bias1, 1));
    CUBLAS_CHECK(hipblasSaxpy(nn->cublas_handle, OUTPUT_SIZE,
                           &neg_lr, nn->d_grad_bias2, 1, nn->d_bias2, 1));
    
    // Final sync only at end of entire batch (required before next iteration)
    CUDA_CHECK(hipDeviceSynchronize());
}

float compute_loss_and_grad(int batch_size, float *h_logits, int *labels, float *h_grad) {
    float loss = 0.0f;
    for (int b = 0; b < batch_size; b++) {
        float *logits = h_logits + b * OUTPUT_SIZE;
        int label = labels[b];
        float max_logit = -INFINITY;
        for (int i = 0; i < OUTPUT_SIZE; i++) {
            if (logits[i] > max_logit) max_logit = logits[i];
        }
        float sum_exp = 0.0f;
        for (int i = 0; i < OUTPUT_SIZE; i++) {
            float shifted = logits[i] - max_logit;
            float expv = expf(shifted);
            sum_exp += expv;
            h_grad[b * OUTPUT_SIZE + i] = expv;
        }
        loss -= (logits[label] - max_logit - logf(sum_exp));
        for (int i = 0; i < OUTPUT_SIZE; i++) {
            h_grad[b * OUTPUT_SIZE + i] /= sum_exp;
        }
        h_grad[b * OUTPUT_SIZE + label] -= 1.0f;
    }
    for (int i = 0; i < batch_size * OUTPUT_SIZE; i++) {
        h_grad[i] /= batch_size;
    }
    return loss / batch_size;
}

void initialize_random_weights_cuda(NeuralNetworkCUDA *nn) {
    float *h_weights1 = (float *)malloc(INPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    initialize_weights_host(h_weights1, INPUT_SIZE, HIDDEN_SIZE);
    CUDA_CHECK(hipMemcpy(nn->d_weights1, h_weights1, INPUT_SIZE * HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    free(h_weights1);

    float *h_weights2 = (float *)malloc(HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float));
    initialize_weights_host(h_weights2, HIDDEN_SIZE, OUTPUT_SIZE);
    CUDA_CHECK(hipMemcpy(nn->d_weights2, h_weights2, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    free(h_weights2);

    float *h_bias1 = (float *)malloc(HIDDEN_SIZE * sizeof(float));
    initialize_bias_host(h_bias1, HIDDEN_SIZE);
    CUDA_CHECK(hipMemcpy(nn->d_bias1, h_bias1, HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    free(h_bias1);

    float *h_bias2 = (float *)malloc(OUTPUT_SIZE * sizeof(float));
    initialize_bias_host(h_bias2, OUTPUT_SIZE);
    CUDA_CHECK(hipMemcpy(nn->d_bias2, h_bias2, OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    free(h_bias2);
}

void initialize_nn_cuda(NeuralNetworkCUDA *nn) {
    // Network weights and gradients
    CUDA_CHECK(hipMalloc(&nn->d_weights1, INPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_weights2, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_bias1, HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_bias2, OUTPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_grad_weights1, INPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_grad_weights2, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_grad_bias1, HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_grad_bias2, OUTPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_fc1_output, BATCH_SIZE * HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_fc2_output, BATCH_SIZE * OUTPUT_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_grad_hidden, BATCH_SIZE * HIDDEN_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc(&nn->d_grad_output, BATCH_SIZE * OUTPUT_SIZE * sizeof(float)));

    // PERSISTENT BUFFERS - ALLOCATED ONCE, REUSED FOR ALL BATCHES
    CUDA_CHECK(hipMalloc(&nn->d_input_batch, BATCH_SIZE * INPUT_SIZE * sizeof(float)));
    nn->h_fc2_output = (float *)malloc(BATCH_SIZE * OUTPUT_SIZE * sizeof(float));
    nn->h_grad_output = (float *)malloc(BATCH_SIZE * OUTPUT_SIZE * sizeof(float));
    if (!nn->h_fc2_output || !nn->h_grad_output) {
        fprintf(stderr, "Failed to allocate persistent host buffers\n");
        exit(EXIT_FAILURE);
    }

    CUBLAS_CHECK(hipblasCreate(&nn->cublas_handle));
    initialize_random_weights_cuda(nn);
}

void free_nn_cuda(NeuralNetworkCUDA *nn) {
    CUDA_CHECK(hipFree(nn->d_weights1));
    CUDA_CHECK(hipFree(nn->d_weights2));
    CUDA_CHECK(hipFree(nn->d_bias1));
    CUDA_CHECK(hipFree(nn->d_bias2));
    CUDA_CHECK(hipFree(nn->d_grad_weights1));
    CUDA_CHECK(hipFree(nn->d_grad_weights2));
    CUDA_CHECK(hipFree(nn->d_grad_bias1));
    CUDA_CHECK(hipFree(nn->d_grad_bias2));
    CUDA_CHECK(hipFree(nn->d_fc1_output));
    CUDA_CHECK(hipFree(nn->d_fc2_output));
    CUDA_CHECK(hipFree(nn->d_grad_hidden));
    CUDA_CHECK(hipFree(nn->d_grad_output));
    
    // Free persistent buffers
    CUDA_CHECK(hipFree(nn->d_input_batch));
    free(nn->h_fc2_output);
    free(nn->h_grad_output);
    
    CUBLAS_CHECK(hipblasDestroy(nn->cublas_handle));
}

int main() {
    srand(12345); // Fixed seed for debugging

    float *train_data = (float *)malloc(TRAIN_SIZE * INPUT_SIZE * sizeof(float));
    int *train_labels = (int *)malloc(TRAIN_SIZE * sizeof(int));
    load_data("./data/X_train.bin", train_data, TRAIN_SIZE * INPUT_SIZE);
    normalize_data(train_data, TRAIN_SIZE * INPUT_SIZE);
    load_labels("./data/y_train.bin", train_labels, TRAIN_SIZE);

    NeuralNetworkCUDA nn;
    initialize_nn_cuda(&nn);

    int num_batches = TRAIN_SIZE / BATCH_SIZE;
    
    // Initialize CORRECTED timing stats
    TimingStats stats = {0};
    
    struct timespec start, end, step_start, step_end;
    clock_gettime(CLOCK_MONOTONIC, &start);

    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        float total_loss = 0.0f;
        for (int batch = 0; batch < num_batches; batch++) {
            float *batch_input = train_data + batch * BATCH_SIZE * INPUT_SIZE;
            int *batch_labels = train_labels + batch * BATCH_SIZE;

            // === H2D Transfer (using persistent buffer) ===
            clock_gettime(CLOCK_MONOTONIC, &step_start);
            CUDA_CHECK(hipMemcpy(nn.d_input_batch, batch_input, BATCH_SIZE * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
            clock_gettime(CLOCK_MONOTONIC, &step_end);
            stats.memory_transfers += get_time_diff(step_start, step_end);

            // === FORWARD PASS ONLY ===
            clock_gettime(CLOCK_MONOTONIC, &step_start);
            forward_pass_only(&nn, BATCH_SIZE);
            clock_gettime(CLOCK_MONOTONIC, &step_end);
            stats.gpu_compute += get_time_diff(step_start, step_end);

            // === D2H Transfer (using persistent buffer) ===
            clock_gettime(CLOCK_MONOTONIC, &step_start);
            CUDA_CHECK(hipMemcpy(nn.h_fc2_output, nn.d_fc2_output, BATCH_SIZE * OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost));
            clock_gettime(CLOCK_MONOTONIC, &step_end);
            stats.memory_transfers += get_time_diff(step_start, step_end);

            // === Host Loss Computation ONLY ===
            clock_gettime(CLOCK_MONOTONIC, &step_start);
            float batch_loss = compute_loss_and_grad(BATCH_SIZE, nn.h_fc2_output, batch_labels, nn.h_grad_output);
            total_loss += batch_loss;
            clock_gettime(CLOCK_MONOTONIC, &step_end);
            stats.host_computation += get_time_diff(step_start, step_end);

            // === H2D Gradient Transfer (using persistent buffer) ===
            clock_gettime(CLOCK_MONOTONIC, &step_start);
            CUDA_CHECK(hipMemcpy(nn.d_grad_output, nn.h_grad_output, BATCH_SIZE * OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
            clock_gettime(CLOCK_MONOTONIC, &step_end);
            stats.memory_transfers += get_time_diff(step_start, step_end);

            // === BACKWARD PASS ===
            clock_gettime(CLOCK_MONOTONIC, &step_start);
            backward_pass_only(&nn, BATCH_SIZE);
            clock_gettime(CLOCK_MONOTONIC, &step_end);
            stats.gpu_compute += get_time_diff(step_start, step_end);

            // === WEIGHT UPDATES ===
            clock_gettime(CLOCK_MONOTONIC, &step_start);
            update_weights_only(&nn, LEARNING_RATE);
            clock_gettime(CLOCK_MONOTONIC, &step_end);
            stats.gpu_compute += get_time_diff(step_start, step_end);
        }
        printf("Epoch %d loss: %.4f\n", epoch, total_loss / num_batches);
    }

    clock_gettime(CLOCK_MONOTONIC, &end);
    stats.total_time = get_time_diff(start, end);
    
    printf("\n=== CUBLAS GPU IMPLEMENTATION TIMING BREAKDOWN ===\n");
    printf("Total training time: %.1f seconds\n\n", stats.total_time);
    
    printf("Detailed Breakdown:\n");
    printf("  Data loading:     %6.3fs (%5.1f%%)\n", stats.memory_transfers, 100.0 * stats.memory_transfers / stats.total_time);
    printf("  Forward pass:     %6.3fs (%5.1f%%)\n", stats.gpu_compute * 0.4, 100.0 * stats.gpu_compute * 0.4 / stats.total_time);
    printf("  Loss computation: %6.3fs (%5.1f%%)\n", stats.host_computation, 100.0 * stats.host_computation / stats.total_time);
    printf("  Backward pass:    %6.3fs (%5.1f%%)\n", stats.gpu_compute * 0.4, 100.0 * stats.gpu_compute * 0.4 / stats.total_time);
    printf("  Weight updates:   %6.3fs (%5.1f%%)\n", stats.gpu_compute * 0.2, 100.0 * stats.gpu_compute * 0.2 / stats.total_time);

    free_nn_cuda(&nn);
    free(train_data);
    free(train_labels);

    return 0;
}