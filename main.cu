// This is for the Test CUDA
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void add1(int*x){
    x[0]+=1;
}

int main() {
    int h = 41, *d = nullptr;
    hipError_t err = hipMalloc(&d, sizeof(int));
    if (err != hipSuccess) { printf("hipMalloc error: %s\n", hipGetErrorString(err)); return 1; }

    hipMemcpy(d, &h, sizeof(int), hipMemcpyHostToDevice);
    add1<<<1,10>>>(d);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) { printf("kernel error: %s\n", hipGetErrorString(err)); return 1; }

    hipMemcpy(&h, d, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d);

    printf("answer = %d\n", h);  // 42 기대
    return 0;
}